#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include "kernels.h"

/* This is your own kernel, you should decide which parameters to add
   here*/
void run_kernel5(const int8_t *filter, int32_t dimension, const int32_t *input,
                 int32_t *output, int32_t width, int32_t height) {
  // Figure out how to split the work into threads and call the kernel below.
}

__global__ void kernel5() {}

__global__ void normalize5() {}
