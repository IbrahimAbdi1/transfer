#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include "kernels.h"





void run_kernel1(const int8_t *filter, int32_t dimension, const int32_t *input,
                 int32_t *output, int32_t width, int32_t height) {
  // Figure out how to split the work into threads and call the kernel below.

  int pixelCount = height*width;
  int32_t *g_min,*g_max;
  int32_t min = 0, max = 255;
  hipMalloc(&g_min,sizeof(int32_t));
  hipMalloc(&g_max,sizeof(int32_t));
  hipMemcpy(g_min,&min,sizeof(int32_t),hipMemcpyHostToDevice);
  hipMemcpy(g_max,&max,sizeof(int32_t),hipMemcpyHostToDevice);

  kernel1<<<pixelCount/1024 + 1,1024>>>(filter,dimension,input,output,width,height,g_min,g_max);
  normalize1<<<pixelCount/1024 + 1,1024>>>(output,width,height,*g_min,*g_max);
   
}

__global__ void kernel1(const int8_t *filter, int32_t dimension, const int32_t *input, 
int32_t *output, int32_t width,int32_t height,int32_t *g_min,int32_t *g_max) {

  // get index given tid
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // call apply2d on input @ index and store it  on output @ index
  if(idx < height*width){
    int row = idx/width;
    int column = idx%width;
   
    // apply2d function
    int32_t sum = 0;
    int filter_centre = dimension/2;
    
    int s_row = row - filter_centre;
    int s_column = column - filter_centre;
    for(int r = 0;r<dimension;r++){
        int n_row = s_row + r;
        for(int c = 0;c<dimension;c++){
            int n_column = s_column + c;
            if((n_row >= 0) && (n_column >= 0) && (n_column < width) && (n_row < height)){
                sum += (filter[r*dimension + c]) * (input[n_row*width + n_column]);
                
            }
        }
    }



    output[idx] = sum;

    if(sum < *(g_min)){
      *g_min = sum;
    }
    if(sum > *(g_max)){
      *g_max = sum;
    }

  }

                          
}

__global__ void normalize1(int32_t *image, int32_t width, int32_t height,
                           int32_t smallest, int32_t biggest) {

  // reduction memes 
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(smallest != biggest){
    image[idx] = ((image[idx] - smallest) * 255) / (biggest - smallest);
  }
}



