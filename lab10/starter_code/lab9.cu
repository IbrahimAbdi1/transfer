#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define M 1024 * 1024
#define threads_block 512
#define MAX_ARR_SIZE 32

#define RUN_SIMPLE 1
#define RUN_THREADS 2
#define RUN_BLOCKS 3
#define RUN_TIMES 4

// Simple kernel: blocks, threads = 1, 1
__global__ void array_add_simple(float *a, float *b, int N) {
  for(int i = 0; i<N;i++){
    a[i] += b[i];
  }
}

// Simple kernel: blocks, threads = 1, 512
__global__ void array_add_threads_only(float *a, float *b, int N) {
  // Edit me!
}

// Complex kernel, utilize both blocks and threads
__global__ void array_add_threads_blocks(float *a, float *b, int N) {
  // Edit me!
}

// Complex kernel, utilize both blocks and threads
// Add b elements 'times' number of times
__global__ void array_add_times(float *a, float *b, int N, int times) {
  // Edit me!
}

/*Initialize the device arrays, timing variables, call kernels
  with the right number of threads and blocks
 */
void run_test(int arrsize, int times, int type) {
  float *a_h, *b_h,*a_d,*b_d;
  hipEvent_t start, stop;
  float transfer_in, computation_time, transfer_out; // timing values
  int N = arrsize * M;

  //dim3 threads(threads_block, 1);
	dim3 blocks((N+threads_block-1)/threads_block);

  size_t SIZE = N * sizeof(float);
  a_h = (float *)malloc(SIZE);
  b_h = (float *)malloc(SIZE);
  hipMalloc((void **)&a_d, SIZE);
  hipMalloc((void **)&b_d, SIZE);

  srand(time(NULL));
  for (int i = 0; i < N; i++) {
    a_h[i] = (rand() % 10000) / 100.0f;
    b_h[i] = (rand() % 10000) / 100.0f;
  }
  hipEventCreate(&start);
	hipEventCreate(&stop);
  hipEventRecord(start);
  hipMemcpy(a_d, a_h, SIZE, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, SIZE, hipMemcpyHostToDevice);
  hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&transfer_in, start, stop);



  if (type == RUN_SIMPLE) {
    hipEventRecord(start);
    array_add_simple <<<1,1>>>(a_d,b_d,N);
  } else if (type == RUN_THREADS) {
    hipEventRecord(start);
    
  } else if (type == RUN_BLOCKS) {
    hipEventRecord(start);
  } else if (type == RUN_TIMES) {
    hipEventRecord(start);
  } else {
    printf("Unknown run type\n");
    goto transfer_out;
  }

  hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&computation_time, start, stop);
transfer_out:
  hipEventRecord(start);
	hipMemcpy(a_h, a_d, SIZE, hipMemcpyDeviceToHost);
  hipMemcpy(b_h, b_d, SIZE, hipMemcpyDeviceToHost);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&transfer_out, start, stop);
  // print timing results. Do not change this printf.
  printf("%5d %5d %15.2f %15.2f %15.2f\n", times, arrsize, transfer_in,
         computation_time, transfer_out);
  hipFree(a_d);
  hipFree(b_d);
  free(a_h);
  free(b_h);
}

int main(int argc, char *argv[]) {
  int arrsize = 1;
  int i;

  // Run with blocks, threads = 1,1
  // Number of times is constant(once), array size varies
  // Do not change this printf.
  printf("Times Size(M) TransferIn(ms) Computation(ms) TransferOut(ms)\n");
  for (arrsize = 1; arrsize <= MAX_ARR_SIZE; i++, arrsize *= 2) {
    run_test(arrsize, 1, RUN_SIMPLE);
  }

  // Run with several blocks and threads
  // Number of times is constant(once), array size varies
  // Do not change this printf.
  printf("\nTimes Size(M) TransferIn(ms) Computation(ms) TransferOut(ms)\n");
  for (arrsize = 1; arrsize <= MAX_ARR_SIZE; i++, arrsize *= 2) {
    run_test(arrsize, 1, RUN_THREADS);
  }

  // Run with several blocks and threads
  // Number of times is constant(once), array size varies
  // Do not change this printf.
  printf("\nTimes Size(M) TransferIn(ms) Computation(ms) TransferOut(ms)\n");
  for (arrsize = 1; arrsize <= MAX_ARR_SIZE; i++, arrsize *= 2) {
    run_test(arrsize, 1, RUN_BLOCKS);
  }

  // Number of times varies, array size is constant (maximum number of elem)
  // Do not change this printf.
  printf("\nTimes Size(M) TransferIn(ms) Computation(ms) TransferOut(ms)\n");
  int times = 1;
  arrsize = MAX_ARR_SIZE;
  for (i = 0; i < 10; i++, times *= 2) {
    run_test(arrsize, times, RUN_TIMES);
  }
}
