
#include <hip/hip_runtime.h>
/* ------------
 * This code is provided solely for the personal and private use of 
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited. 
 * All forms of distribution of this code, whether as given or with 
 * any changes, are expressly prohibited. 
 * 
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 * 
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
*/

__inline__ __device__
int warpReduceSum(int val)
{
    for (int offset = warpSize/2; offset > 0; offset /= 2) 
    {
        val += __shfl_down(val, offset);
    }
    return val;
}

__inline__ __device__
int blockReduceSum(int val)
{
    static __shared__ int shared[32]; // Shared mem for 32 partial sums
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    val = warpReduceSum(val);     // Each warp performs partial reduction

    if (lane==0) shared[wid]=val; // Write reduced value to shared memory

    __syncthreads();              // Wait for all partial reductions

    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

    if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

    return val;
}

__global__ void reduce8(int *in, int* out, unsigned int N)
{
    int sum = 0;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; 
             i += blockDim.x * gridDim.x)
    {
        sum += in[i];
    }

    sum = blockReduceSum(sum);
    if (threadIdx.x==0)
    {
        out[blockIdx.x]=sum;
    }
}

__global__ void reduce9(int *in, int* out, unsigned int N)
{
    int sum = 0;
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
            i += blockDim.x * gridDim.x)
    {
        sum += in[i];
    }

    sum = warpReduceSum(sum);
    if ((threadIdx.x & (warpSize - 1)) == 0)
        atomicAdd(out, sum);
}

__global__ void reduce10(int *in, int* out, unsigned int N)
{
    int sum = 0;
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; 
            i < N; 
            i += blockDim.x * gridDim.x) {
        sum += in[i];
    }
    sum = blockReduceSum(sum);
    if (threadIdx.x == 0)
        atomicAdd(out, sum);
}
