#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of 
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited. 
 * All forms of distribution of this code, whether as given or with 
 * any changes, are expressly prohibited. 
 * 
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 * 
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2019 Bogdan Simion
 * -------------
*/

#include "kernels.h"

__global__ void reduce4(int *g_idata, int *g_odata) {

	extern __shared__ int sdata[];
	
	unsigned int tid = threadIdx.x;
	
	// Global thread id
	unsigned int i = blockIdx.x * (blockDim.x*2) + threadIdx.x;

	sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
	__syncthreads();

    // do reduction in shared memory
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) { 
        if (tid < s) {  
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
	// write result for this block back to global memory
	if (tid == 0) { g_odata[blockIdx.x] = sdata[0]; }
}
